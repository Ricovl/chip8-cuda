#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <thrust/count.h>
#include <thrust/device_ptr.h>
#include <thrust/execution_policy.h>

#include <hip/hip_runtime_api.h>
#include <iostream>


#define PROGRAM_START 0x200

typedef struct chip8_t {
    uint8_t mem[4096];

    uint16_t pc;
    uint16_t sp;
    uint16_t I;

    uint8_t v[16];

    uint8_t vram[64 * 32];
};

__device__ bool chip8_step(chip8_t *chip, uint8_t *program) {
    uint8_t x_start, y_start, height;
    uint16_t opcode = chip->mem[chip->pc] << 8 | chip->mem[chip->pc + 1];
    // uint8_t tmp = (opcode & 0xff00) >> 8;
    // opcode = ((opcode & 0x00ff) << 8) | tmp;

    printf("[%d] ", chip->pc);

    uint8_t inst = (uint8_t)((opcode & 0xf000) >> 12);
    switch (inst)
    {
    case 0x0:
        switch (opcode)
        {
        case 0x00E0:
            printf("disp_clear()\n");
            memset(chip->vram, 0, 64*32);
            break;
        case 0x00EE:
            printf("return;\n");
            break;
        default:
            printf("Unknown instruction: %04X %d\n", opcode, inst);
            break;
        }

        break;
    case 0x1:
        printf("goto %d;\n", ((opcode & 0x0fff) >> 0)); 
        chip->pc = opcode & 0x0fff;
        chip->pc -= 2;

        break;
    // case 0x2:
    //     break;
    // case 0x3:

    //     break;
    // case 0x4:

    //     break;
    // case 0x5:

    //     break;
    case 0x6:
        printf("V%d = %d;\n", (opcode & 0x0f00) >> 8, ((opcode & 0x00ff) >> 0)); 
        chip->v[(opcode & 0x0f00) >> 8] = (opcode & 0x00ff) >> 0;
        break;
    case 0x7:
        printf("V%d += %d;\n", (opcode & 0x0f00) >> 8, ((opcode & 0x00ff) >> 0)); 
        chip->v[(opcode & 0x0f00) >> 8] += (opcode & 0x00ff) >> 0;
        break;
    // case 0x8:

    //     break;
    // case 0x9:

    //     break;
    case 0xA:
        printf("I = %d;\n", (opcode & 0x0fff) >> 0);
        chip->I = opcode & 0x0fff;
        break;
    // case 0xB:

    //     break;
    // case 0xC:

    //     break;
    case 0xD:
        printf("draw(V%d, V%d, %d);\n", (opcode & 0x0f00) >> 8, (opcode & 0x00f0) >> 4, (opcode & 0x000f) >> 0); 
        printf("draw(%d, %d, %d);\n", chip->v[(opcode & 0x0f00) >> 8], chip->v[(opcode & 0x00f0) >> 4], (opcode & 0x000f) >> 0); 
        x_start = chip->v[(opcode & 0x0f00) >> 8];
        y_start = chip->v[(opcode & 0x00f0) >> 4];
        height = (opcode & 0x000f) >> 0;

        chip->v[0xF] = 0;
        for (int y = 0; y < height; y++) {
            uint8_t pixel = chip->mem[chip->I + y];

            for (int x = 0; x < 8; x++) {
                if ((pixel & (0x80 >> x)) != 0) {
                    if (chip->vram[((y_start + y) * 64) + x_start + x] == 1) {
                        chip->v[0xF] = 1;
                    }
                    chip->vram[((y_start + y) * 64) + x_start + x] ^= 1;
                    printf("xor");
                }
            }
        }

        break;
    // case 0xE:

    //     break;
    // case 0xF:

    //     break;
    default:
        printf("Unknown instruction: %04X %d\n", opcode, inst);
        break;
    }

    chip->pc += 2;
    return true;
}

__global__ void run_chip8(chip8_t *chip, uint8_t *program, int n)
{
    // const char letters[]{'x', 'y', 'z', 'w'};
    printf("test %d, %d, %d\n", blockIdx.x, threadIdx.x, (blockIdx.x * blockDim.x) + threadIdx.x);
    chip8_t *local_chip = &chip[(((blockIdx.x * blockDim.x) + threadIdx.x))];
    unsigned i = 0;

    local_chip->pc = PROGRAM_START;
    memcpy(local_chip->mem + PROGRAM_START, program, n);

    bool step = true;
    while (step && i < 30) {
        step = chip8_step(local_chip, program);

        if (local_chip->pc > n + PROGRAM_START) {
            step = false;
        }
        i++;
    }


}

// #define NUM_BLOCKS 4
// #define NUM_THREADS_PER_BLOCK 8
#define NUM_BLOCKS 2
#define NUM_THREADS_PER_BLOCK 8

int main(int argc, char **argv)
{
    const char *filename = sdkFindFilePath("IBM Logo.ch8", argv[0]);

    // find first CUDA device
    int devID = findCudaDevice(argc, (const char **)argv);

    std::ifstream file(filename, std::ios::binary | std::ios::ate);
    std::streamsize len = file.tellg();
    file.seekg(0, std::ios::beg);

    char *program;
    program = (char *)malloc(len);
    if (!file.read(program, len)) {
        printf("Cannot find the input text file\n. Exiting..\n");
        return EXIT_FAILURE;
    }
    file.close();
    std::cout << "Read " << len << " byte corpus from " << filename << std::endl;

    uint8_t *d_program;
    checkCudaErrors(hipMalloc(&d_program, len));
    checkCudaErrors(hipMemcpy(d_program, program, len, hipMemcpyHostToDevice));

    // allocate 4kb for each emulator
    chip8_t *d_chip;
    chip8_t *h_chip = (chip8_t *)malloc(sizeof(chip8_t) * (NUM_BLOCKS * NUM_THREADS_PER_BLOCK));
    checkCudaErrors(hipMalloc(&d_chip, sizeof(chip8_t) * (NUM_BLOCKS * NUM_THREADS_PER_BLOCK)));


    // Try uncommenting one kernel call at a time
    run_chip8<<<NUM_BLOCKS, NUM_THREADS_PER_BLOCK>>>(d_chip , d_program, len);
    checkCudaErrors(hipMemcpy(h_chip, d_chip, sizeof(chip8_t) * (NUM_BLOCKS * NUM_THREADS_PER_BLOCK), hipMemcpyDeviceToHost));

    for (int i = 0; i < NUM_BLOCKS * NUM_THREADS_PER_BLOCK; i++) {
        printf("output for %d:\n", i);
        for (int y = 0; y < 32; y++) {
            for (int x = 0; x < 64; x++) {
                printf("%d", h_chip[i].vram[(y * 64) + x]);
            }
            printf("\n");
        }
    }

    checkCudaErrors(hipFree(d_program));
    checkCudaErrors(hipFree(d_chip));

    return EXIT_SUCCESS;
}
